#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdbool.h>

#define NUM_THREADS 4
bool printout = false;

/********************** HOST FUNCTION DECLARATIONS ****************************/
void invalid_args(int error);
int* get_rand_array(int size);
int **get_rand_square_matrix(int size);
int* flatten2Dmatrix(int **matrix, int size);
void print_array(int* array, int size);
void print_matrix(int **matrix, int size);
int* merge(int* arrayParts, int size, int part);


/********************** DEVICE FUNCTIONS *************************************/

__global__ void return_first(int *array, int *result) {
   if (threadIdx.x == 0)
      atomicExch(result, array[0]);
   __syncthreads();
}

__global__ void linear_search(int *array, int *N, int *M, int *key, int *result) {
   int start = *M * threadIdx.x;
   int end = *N < start + *M ? *N : start + *M;   
   for (int i = start; i < end; i++)
      if (array[i] == *key)
         atomicExch(result, i);
   __syncthreads();
}

__global__ void bubble_sort(int *array, int *N, int *M) {
   int i, j, temp;
   int start = *M * threadIdx.x;
   int end = *N < start + *M ? *N : start + *M;   
   //printf("\nThread %d starts at index %d and ends at index %d", threadIdx.x, start, end);
   for (i = start; i < end; i++) {
      for (j = start; j < start+end-i-1; j++)
         if (array[j] > array[j + 1]) {
            temp = array[j];
            array[j] = array[j+1];
            array[j+1] = temp;
         }
   }
   __syncthreads();
}
__global__ void matrix_multiplication(int *d_flat_matrix, int *N, int *M, int *d_results) {
   int start = threadIdx.x * *M * *N;
   int end = *N * *N < start + *M * *N ? *N * *N : start + *M * *N;
   //printf("\nThread %d starts at index %d and ends at index %d", threadIdx.x, start, end);
   for (int i = start, j = 0; i < end; i += *N, j++)
      for (int k = 0; k < *N; k++)
         for (int l = 0; l < *N; l++)
            d_results[start + j * *N + k] += d_flat_matrix[i + l] * d_flat_matrix[l * *N + k];
   __syncthreads();
}

int main(int argc, char *argv[]) {

   // Declare host variables
   int option;       // the option/function to execute
   int N;            // the data dimension (N,1) for array & (N,N) for matrix
   int* array;       // the host's array of ints
   int** matrix;     // host copy of the square matrix
   int* flat_matrix; // the matrix, but flattened
   int size;         // the total amount of data
   int num_blocks;   // the number of CUDA blocks to use
   int num_threads = NUM_THREADS;  // the number of CUDA threads to use
   int M;            // the number of data items per thread
   int key;          // the key to find if doing linear search
   int result;       // single int result of return first and linear search

   // Declare device variables
   int* d_N;      // device copy of array size
   int* d_array;  // device copy of array
   int* d_M;      // device copy of elements per thread
   int* d_key;    // device copy of key if linear searching
   int* d_flat_matrix;// device copy of flattened 2D matrix
   int* d_result;  // device copy of single digit results
   int* d_results; // device copy of the result of matmul

   // Declare the CUDA timers
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   // Process the given CLI arguments and process any errors
	int error = -1;
	if (argc != 3) 
		error = 0;  // invalid arg count
	else {
		option = atoi(argv[1]);
		N = atoi(argv[2]);
	}
	if (N == 0 || option > 3 || option < 0) 
		error = 1;  // invalid arg vals
	if (error != -1){
      invalid_args(error);
		exit(0);
	}

   // Initialize host variables
   size = N * sizeof(int);
   num_blocks = 1;
   M = ceil(1.0 * N / num_threads); // elements (or # rows) per thread

   // Allocate space on device for the host's option and data
   hipMalloc((void **)&d_N, sizeof(int));
   hipMalloc((void **)&d_M, sizeof(int));
   if (option == 3) { 
      hipMalloc((void **)&d_flat_matrix, size * size);
      hipMalloc((void **)&d_results, size * size);
      hipMemset((void **)&d_results, 0, size * size);
   } else
      hipMalloc((void **)&d_array, size);
   if (option == 0 || option == 1)
      hipMalloc((void **)&d_result, sizeof(int));
   if (option == 1)
      hipMalloc((void **)&d_key, sizeof(int));

   // Initialize the host's random data
   if (option == 3) {
      matrix = get_rand_square_matrix(N);
      if (printout) {
         printf("\nOriginal Matrix:\n");
         print_matrix(matrix, N);
      }
      flat_matrix = (int *) malloc(N * size);
      flat_matrix = flatten2Dmatrix(matrix, N);
   } else {
      array = get_rand_array(N);
      if (printout) {
         printf("\nOriginal Array:\n");
         print_array(array, N);
      }
      if (option == 1)
         key = array[rand() % N];
   }

   // Start timer to record total data transfer and function execution time
   hipEventRecord(start, 0);

   // Copy host array, length, and elements per thread to device
   hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(d_M, &M, sizeof(int), hipMemcpyHostToDevice);
   if (option == 3) {
      hipMemcpy(d_flat_matrix, flat_matrix, size * size, hipMemcpyHostToDevice);
   } else {
      hipMemcpy(d_array, array, size, hipMemcpyHostToDevice);
      if (option == 1)
         hipMemcpy(d_key, &key, sizeof(int), hipMemcpyHostToDevice);
   }

   // Execute option and copy result back to the host
   switch (option) {
      case 0:
         return_first<<<num_blocks, num_threads>>>(d_array, d_result);
         hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
         break;
      case 1:
         linear_search<<<num_blocks, num_threads>>>(d_array, d_N, d_M, d_key, d_result);
         hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
         break;
      case 2:
         bubble_sort<<<num_blocks, num_threads>>>(d_array, d_N, d_M);
         hipMemcpy(array, d_array, size, hipMemcpyDeviceToHost);
         break;
      case 3:
         matrix_multiplication<<<num_blocks, num_threads>>>(d_flat_matrix, d_N, d_M, d_results);
         hipMemcpy(flat_matrix, d_results, size * size, hipMemcpyDeviceToHost);
         break;
   }

   // Merge sorta-sorted array
   if (option == 2) {
      if (printout) {
         printf("\nSorta-sorted Array:\n");
         print_array(array, N);
      }   
      array = merge(array, N, M);
   }

   // end timer and report execution time
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   float elapsed_time;
   hipEventElapsedTime(&elapsed_time, start, stop);
   hipEventDestroy(start); hipEventDestroy(stop);
   printf("\nExecution took %f ms", elapsed_time);

   // display results
   if (printout) {
      switch (option) {
         case 0:
            printf("\nThread 0 returned the first item in the array: %d", result);
            break;
         case 1:
            printf("\nSearch key %d found at index %d", key, result);
            break; // TODO calculate and report which thread found the key
         case 2:
            printf("\nSorted Array:\n");
            print_array(array, N);      
            break;
         case 3:
            printf("\nFinal Matrix result:\n");
            for (int i = 0; i < N * N; i++) {
               if (i % N == 0) {
                  print_array(&flat_matrix[i], N);
                  printf("\n");
               }
            }
            break;
      }
   }

   // Cleanup global CUDA variables
   if (option == 3)
      printf("\nError freeing memory on GPU (Result unaffected):\n");
   hipFree(d_N); hipFree(d_M);
   if (option == 3) {
      hipFree(d_flat_matrix);
      hipFree(d_results);
   } else {
      hipFree(d_array);
      if (option == 1)
         hipFree(d_key);
      if (option == 0 || option == 1)
         hipFree(d_result);    
   }

   return 0;
}

/********************** HOST FUNCTIONS *************************************/
void invalid_args(int error) {
	switch(error){
		case 0: printf("Error: Invalid number of arguments.\n"); break;
		case 1: printf("Error: Invalid argument values.\n"); break;
	}
	printf("Usage: <option> <data dimension>\n");
	printf("Valid options are:\n \
		0) Return First\n \
		1) Linear Search\n \
		2) Bubble Sort\n \
		3) Matrix Multiplication\n");
	printf("Valid data dimension must be greater than 0 and the number of processors.\n");
}

int* get_rand_array(int size) {
    srand((unsigned)time(NULL));    
    int* array = (int*) malloc(size * sizeof(int));
    for (int i = 0; i < size; i++)
        array[i] = rand() % 100000;
    return array;
}

int **get_rand_square_matrix(int size){
   srand((unsigned)time(NULL));
   int **matrix = (int **)malloc(sizeof(int*) * size);
   for(int i = 0; i < size; i++)
       matrix[i] = (int *)malloc(sizeof(int) * size);
   for (int i = 0; i < size; i++)
       for (int j = 0; j < size; j++)
           matrix[i][j] = rand() % 10000;
   return matrix;
}

int* flatten2Dmatrix(int **matrix, int size) {
	int *flat_matrix = (int *)malloc(size*size*sizeof(int));
	for (int i = 0; i < size; i++)
		for (int j = 0; j < size; j++)
			flat_matrix[i * size + j] = matrix[i][j];
	return flat_matrix;
}

void print_array(int* array, int size) {
   for (int i = 0; i < size; i++)
      printf("%d ",*(array + i));
}

void print_matrix(int **matrix, int size) {
   for (int i = 0; i < size; i++){
       for (int j = 0; j < size; j++)
           printf("%d ",matrix[i][j]);
       printf("\n");
   }
}

int* merge(int* arrayParts, int size, int part) {
	int* result = (int*) calloc(size, sizeof(int));
	int min_value = INT_MAX;
	int min_index;
	int removed = 0;
	while (removed < size) {
      // search for the min item at the front of each array part
		for (int i = 0; i < size; i += part) {
			if (arrayParts[i] != -1 && arrayParts[i] < min_value) {
				min_value = arrayParts[i];
				min_index = i;
			}
		}
		// shift everything else in the array left until part is reach
		int j; // could stop shifting once a -1 is reached...
		for (j = min_index; j < size - 1 && j < min_index + part - 1; j++)
			arrayParts[j] = arrayParts[j + 1];
		arrayParts[j] = -1; // set empties to -1
		// insert the min item to our new array
		result[removed++] = min_value;
		min_value = INT_MAX;
	}
	return result;
}
